#include "gtest/gtest.h"

#include <memory>

#include "hip/hip_runtime.h"
#include ""

TEST(UniquePtrTest, TestCudaMemcpy) {
  auto n = 100000;

  std::unique_ptr<int[]> foo(new int[n]);
  for (auto i = 0; i < n; ++i) {
    foo[i] = i;
  }

  int *bar;
  hipMallocManaged(&bar, n * sizeof(int));
  hipMemcpy(bar, foo.get(), n * sizeof(int), hipMemcpyHostToDevice);

  for (auto i = 0; i < n; ++i) {
    foo[i] = 0;
  }

  // establish preconditions for the test
  for (auto i = 0; i < n; ++i) {
    EXPECT_EQ(0, foo[i]);
  }

  // copy back to unique_ptr
  hipMemcpy(foo.get(), bar, n * sizeof(int), hipMemcpyDeviceToHost);

  // establish preconditions for the test
  for (auto i = 0; i < n; ++i) {
    EXPECT_EQ(i, foo[i]);
  }

  hipFree(bar);
}
