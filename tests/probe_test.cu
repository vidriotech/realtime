#include "hip/hip_runtime.h"
#include "gtest/gtest.h"

#include <stdexcept>
#include "../src/probe/probe.cuh"
#include "./test_utilities/test_utilities.cuh"

TEST(ProbeTestSuite, TestInitOK) {
  unsigned n_tot = 385, n_active = 384, n_groups = 4;
  double srate_hz = 30000.0;

  ProbeConfig cfg = make_probe_config(n_tot, n_active, n_groups, srate_hz);
  Probe prb(cfg);

  EXPECT_EQ(n_tot, prb.n_total());
  EXPECT_EQ(n_active, prb.n_active());
}

TEST(ProbeTestSuite, TestInitMoreActiveThanTotalFails) {
  unsigned n_tot = 385, n_active = 388, n_groups = 4; // 388 > 385
  double srate_hz = 30000.0;

  ProbeConfig cfg = make_probe_config(n_tot, n_active, n_groups, srate_hz);

  EXPECT_THROW(Probe prb(cfg), std::domain_error);
}

TEST(ProbeTestSuite, TestMakeDistMatrixOK) {
  unsigned n_tot = 4, n_active = 4, n_groups = 2;
  double srate_hz = 30000.0;

  Probe prb = make_probe(n_tot, n_active, n_groups, srate_hz);
  prb.MakeDistanceMatrix();

  EXPECT_EQ(0.0, prb.dist_between(0, 0));
  EXPECT_NEAR(20.0, prb.dist_between(0, 1), 1e-12);
  EXPECT_NEAR(32.0156, prb.dist_between(0, 2), 1e-4);
  EXPECT_NEAR(47.1699, prb.dist_between(0, 3), 1e-4);
  EXPECT_EQ(0.0, prb.dist_between(1, 1));
  EXPECT_NEAR(25.0, prb.dist_between(1, 2), 1e-12);
  EXPECT_NEAR(32.0156, prb.dist_between(1, 3), 1e-4);
  EXPECT_EQ(0.0, prb.dist_between(2, 2));
  EXPECT_NEAR(20.0, prb.dist_between(2, 3), 1e-12);
  EXPECT_EQ(0.0, prb.dist_between(3, 3));
}

/*
 * GIVEN a Probe probe
 * TEST THAT each active channel is actually reported as active.
 */
TEST(ProbeTestSuite, IsActive) {
  auto n_channels = std::stoi(get_env_var("TEST_NCHANNELS"));
  auto n_active = std::stoi(get_env_var("TEST_NACTIVE"));
  auto probe = probe_from_env();

  for (auto i = 0; i < n_active; ++i) {
    EXPECT_TRUE(probe.is_active(i));
  }

  for (auto i = n_active; i < n_channels; i++) {
    EXPECT_FALSE(probe.is_active((i)));
  }
}

/*
 * GIVEN a Probe `probe`, a site index `site_idx`, and a positive integer `n`
 * TEST THAT the distances to its reported n nearest neighbors are sorted; AND
 *           the smallest distance to a site *not* a nearest neighbor is
 *           at least as large as the largest distance of the nearest
 *           neighbors.
 */
TEST(ProbeTestSuite, NearestNeighbors) {
  auto probe = probe_from_env();

  ASSERT_GT(probe.n_active(), 0);
  auto site_idx = std::max((uint32_t) 0, probe.n_active() - 1);
  auto n_neighbors = 10;

  // compute nearest neighbors
  probe.MakeDistanceMatrix();
  auto nearest_neighbors = probe.NearestNeighbors(site_idx, n_neighbors);

  // test that distances given are sorted
  std::vector<float> dists(n_neighbors);
  for (auto i = 0; i < n_neighbors; ++i) {
    dists.at(i) = probe.dist_between(site_idx, nearest_neighbors.at(i));
  }

  EXPECT_TRUE(std::is_sorted(dists.begin(), dists.end()));

  // test that the min distance of sites not in nearest neighbors is no
  // smaller than the largest distance in nearest neighbors
  auto largest_distance = *(dists.end());
  std::vector<uint32_t> non_neighbors;
  for (auto i = 0; i < probe.n_active(); ++i) {
    auto it = std::find(nearest_neighbors.begin(), nearest_neighbors.end(), i);

    // value not found in nearest neighbors; test it
    if (it == nearest_neighbors.end()) {
      EXPECT_GE(probe.dist_between(site_idx, i), largest_distance);
    }
  }
}

/*
 *
 */
TEST(ProbeTestSuite, SiteIndex) {
  auto probe = probe_from_env();

  auto site_idx = 0;
  for (auto chan = 0; chan < probe.n_total(); ++chan) {
    if (probe.is_active(chan)) {
      EXPECT_EQ(site_idx++, probe.site_index(chan));
    }
  }
}