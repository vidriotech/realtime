#include "gtest/gtest.h"
#include "../src/kernels/kernels.cuh"

TEST(KernelTestSuite, TestNdiff2KernelShort) {
  auto nchans = 64;

  short *data, *filtered;
  hipMallocManaged(&data, 4 * nchans * sizeof(short));
  hipMallocManaged(&filtered, 4 * nchans * sizeof(short));

  /*
   * channel values: 1 1 2 2 -> (-1 * 1) + (-2 * 1) + (2 * 2) + (1 * 2) = 3
   */
  for (auto i = 0; i < 4 * nchans; i++) {
    if (i < 2 * nchans) {
      data[i] = 1;
    } else {
      data[i] = 2;
    }

    filtered[i] = 0;
  }

  auto nthreads = 256;
  auto nblocks = (4 * nchans + nthreads - 1) / nthreads;

  ndiff2_<<<nblocks, nthreads>>>(4 * nchans, nchans, data, filtered);
  hipDeviceSynchronize();

  /*
   * filtered values at indices 0, 2, and 3 get 0, while channel value at index 1 gets 3
   */
  for (auto i = 0; i < 4 * nchans; i++) {
    EXPECT_EQ((i >= nchans && i < 2 * nchans) ? 3 : 0, filtered[i]);
  }

  // cleanup
  hipFree(data);
  hipFree(filtered);
}

TEST(KernelTestSuite, TestNdiff2KernelFloat) {
  auto nchans = 64;

  float *data, *filtered;
  hipMallocManaged(&data, 4 * nchans * sizeof(float));
  hipMallocManaged(&filtered, 4 * nchans * sizeof(float));

  /*
   * channel values: 1 1 2 2 -> (-1 * 1) + (-2 * 1) + (2 * 2) + (1 * 2) = 3
   */
  for (auto i = 0; i < 4 * nchans; i++) {
    if (i < 2 * nchans) {
      data[i] = 1.0f;
    } else {
      data[i] = 2.0f;
    }

    filtered[i] = 0;
  }

  auto nthreads = 256;
  auto nblocks = (4 * nchans + nthreads - 1) / nthreads;

  ndiff2_<<<nblocks, nthreads>>>(4 * nchans, nchans, data, filtered);
  hipDeviceSynchronize();

  /*
   * filtered values at indices 0, 2, and 3 get 0, while channel value at index 1 gets 3
   */
  for (auto i = 0; i < 4 * nchans; i++) {
    EXPECT_EQ((i >= nchans && i < 2 * nchans) ? 3.0f : 0.0f, filtered[i]);
  }

  // cleanup
  hipFree(data);
  hipFree(filtered);
}

TEST(KernelTestSuite, TestNdiff2Short) {
  auto nchans = 64;

  auto n_frames = 4;
  short *data, *filtered;
  hipMallocManaged(&data, n_frames * nchans * sizeof(short));
  hipMallocManaged(&filtered, n_frames * nchans * sizeof(short));

  /*
   * channel values: 1 1 2 2 -> (-1 * 1) + (-2 * 1) + (2 * 2) + (1 * 2) = 3
   */
  for (auto i = 0; i < n_frames * nchans; i++) {
    if (i < 2 * nchans) {
      data[i] = 1;
    } else {
      data[i] = 2;
    }

    filtered[i] = 0;
  }

  auto nthreads = 256;
  auto nblocks = (n_frames * nchans + nthreads - 1) / nthreads;

  ndiff2(n_frames * nchans, nchans, data, filtered, nblocks, nthreads);

  /*
   * filtered values at indices 0, 2, and 3 get 0, while channel value at index 1 gets 3
   */
  for (auto i = 0; i < n_frames * nchans; i++) {
    EXPECT_EQ((i >= nchans && i < 2 * nchans) ? 3 : 0, filtered[i]);
  }

  // cleanup
  hipFree(data);
  hipFree(filtered);
}

/*
 * GIVEN a buffer `data_` of int16 and a constant detect `const_thresh`
 * TEST THAT values in `data_` which exceed `const_thresh` correspond to true
 *           values in a boolean data `crossings_`.
 */
TEST(KernelTestSuite, FindCrossingsKernelShort) {
  auto n_channels = 100;
  auto n_frames = 100;
  auto n_samples = n_channels * n_frames;
  auto const_thresh = 9.0f;

  short *data;
  uint8_t *crossings;
  float *thresholds;

  hipMallocManaged(&data, n_samples * sizeof(short));
  hipMallocManaged(&crossings, n_samples * sizeof(bool));
  hipMallocManaged(&thresholds, n_channels * sizeof(float));

  for (auto i = 0; i < n_channels; ++i) {
    thresholds[i] = const_thresh;
  }

  // column j gets all j's
  for (auto k = 0; k < n_samples; ++k) {
    data[k] = (short) (-k / n_channels);
  }

  // establish preconditions for the test
  for (auto k = 0; k < n_samples; k++) {
    EXPECT_FALSE(crossings[k]);

    if (k < n_channels * (const_thresh + 1)) {
      EXPECT_FALSE(data[k] < -const_thresh);
    } else {
      EXPECT_TRUE(data[k] < -const_thresh);
    }
  }

  // perform the thresholding
  auto n_threads = 256;
  auto n_blocks = (n_samples + n_threads - 1) / n_threads;
  find_crossings_<<<n_blocks, n_threads>>>(n_samples, n_channels, data,
                                           thresholds, crossings);
  hipDeviceSynchronize();

  // test crossings_ detected correctly
  for (auto k = 0; k < n_samples; k++) {
    if (k < n_channels * (const_thresh + 1)) {
      EXPECT_FALSE(crossings[k]);
    } else {
      EXPECT_TRUE(crossings[k]);
    }
  }

  // clean up
  hipFree(data);
  hipFree(crossings);
  hipFree(thresholds);
}

/*
* GIVEN a buffer `data_` of float32 and a constant detect `const_thresh`
* TEST THAT values in `data_` which exceed `const_thresh` correspond to true
*           values in a boolean data `crossings_`.
*/
TEST(KernelTestSuite, FindCrossingsKernelFloat) {
  auto n_channels = 100;
  auto n_frames = 100;
  auto n_samples = n_channels * n_frames;
  auto const_thresh = 9.0f;

  float *data;
  uint8_t *crossings;
  float *thresholds;

  hipMallocManaged(&data, n_samples * sizeof(float));
  hipMallocManaged(&crossings, n_samples * sizeof(bool));
  hipMallocManaged(&thresholds, n_channels * sizeof(float));

  for (auto i = 0; i < n_channels; ++i) {
    thresholds[i] = const_thresh;
  }

  // column j gets all j's
  for (auto k = 0; k < n_samples; ++k) {
    data[k] = (float) (-k / n_channels); // NOLINT(bugprone-integer-division)
  }

  // establish preconditions for the test
  for (auto k = 0; k < n_samples; k++) {
    EXPECT_FALSE(crossings[k]);

    if (k < n_channels * (const_thresh + 1)) {
      EXPECT_FALSE(data[k] < -const_thresh);
    } else {
      EXPECT_TRUE(data[k] < -const_thresh);
    }
  }

  // perform the thresholding
  auto n_threads = 256;
  auto n_blocks = (n_samples + n_threads - 1) / n_threads;
  find_crossings_<<<n_blocks, n_threads>>>(n_samples, n_channels, data,
                                           thresholds, crossings);
  hipDeviceSynchronize();

  // test crossings_ detected correctly
  for (auto k = 0; k < n_samples; k++) {
    if (k < n_channels * (const_thresh + 1)) {
      EXPECT_FALSE(crossings[k]);
    } else {
      EXPECT_TRUE(crossings[k]);
    }
  }

  // clean up
  hipFree(data);
  hipFree(crossings);
  hipFree(thresholds);
}

/*
 *
 */
TEST(KernelTestSuite, FindCrossingsShort) {
  auto n_channels = 100;
  auto n_frames = 100;
  auto n_samples = n_channels * n_frames;
  auto const_thresh = 9.0f;

  short *data;
  uint8_t *crossings;
  float *thresholds;

  hipMallocManaged(&data, n_samples * sizeof(short));
  hipMallocManaged(&crossings, n_samples * sizeof(bool));
  hipMallocManaged(&thresholds, n_channels * sizeof(float));

  for (auto i = 0; i < n_channels; ++i) {
    thresholds[i] = const_thresh;
  }

  // column j gets all j's
  for (auto k = 0; k < n_samples; ++k) {
    data[k] = (short) (-k / n_channels);
  }

  // establish preconditions for the test
  for (auto k = 0; k < n_samples; k++) {
    EXPECT_FALSE(crossings[k]);

    if (k < n_channels * (const_thresh + 1)) {
      EXPECT_FALSE(data[k] < -const_thresh);
    } else {
      EXPECT_TRUE(data[k] < -const_thresh);
    }
  }

  // perform the thresholding
  auto n_threads = 256;
  auto n_blocks = (n_samples + n_threads - 1) / n_threads;
  find_crossings(n_samples, n_channels, data, thresholds, crossings,
                 n_blocks, n_threads);

  // test crossings_ detected correctly
  for (auto k = 0; k < n_samples; k++) {
    if (k < n_channels * (const_thresh + 1)) {
      EXPECT_FALSE(crossings[k]);
    } else {
      EXPECT_TRUE(crossings[k]);
    }
  }

  // clean up
  hipFree(data);
  hipFree(crossings);
  hipFree(thresholds);
}