#include "hip/hip_runtime.h"
#include "pca.cuh"

/**
 * @brief
 * @param args
 */
void center_features(CenterFeaturesArgs &args) {
  auto n_obs = args.n_obs;
  auto n_feats = args.n_feats;
  auto features = args.features;

  auto transpose_idx =
      thrust::make_transform_iterator(thrust::counting_iterator<int>(0),
                                      transpose(n_feats, n_obs));

  auto features_t =
      thrust::make_permutation_iterator(features.begin(), transpose_idx);

  auto mean_iter =
      thrust::make_transform_iterator(features_t,
                                      mean_functor((float) n_feats));

  auto row_iter =
      thrust::make_transform_iterator(thrust::counting_iterator<int>(0),
                                      idx_to_row_idx(n_feats));

  // allocate storage for row sums and indices
  thrust::device_vector<float> row_means(n_obs);
  thrust::device_vector<int> row_indices(n_obs);

  // compute the means for each column (row in the transposed matrix)
  thrust::reduce_by_key
      (row_iter, row_iter + features.size(),
       mean_iter,
       row_indices.begin(),
       row_means.begin(),
       thrust::equal_to<int>(),
       thrust::plus<float>());

  // subtract the means from each column (row in the transposed matrix)
  // copy back into features array
  thrust::device_ptr<float> means_ptr = row_means.data();
  thrust::transform(features_t,
                    features_t + (n_feats * n_obs),
                    row_iter,
                    features.begin(),
                    mean_subtract(thrust::raw_pointer_cast(means_ptr)));

  // transpose the subtracted matrix back to its original form
  transpose_idx =
      thrust::make_transform_iterator(thrust::counting_iterator<int>(0),
                                      transpose(n_obs, n_feats));

  features_t =
      thrust::make_permutation_iterator(features.begin(), transpose_idx);

  // assign back into arg.features
  args.features.assign(features_t,
                       features_t + (n_obs * n_feats));
}

/**
 * @brief Compute a covariance matrix from a matrix of centered observations.
 *
 * In the observation matrix, it's expected that each row of the matrix is an
 * observation, and since the matrix is stored in row-major order (per C/C++
 * convention), the corresponding array is stored as one observation stacked
 * after another. Moreover, the observations are expected to be centered,
 * i.e., the mean of each feature (namely, each column) should be
 * approximately zero.
 *
 * @param args Struct of arguments.
 */
void make_cov_matrix(CovMatrixArgs &args) {
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  auto n_obs = args.n_obs;
  auto n_feats = args.n_feats;
  auto features = args.features;
  auto cov_matrix = args.cov_matrix;

  auto alpha = 1.0f / ((float) n_obs - 1);
  auto beta = 0.f;

  hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n_feats, n_feats, n_obs, &alpha,
              features, n_obs, features, n_obs, &beta, cov_matrix, n_obs);

  hipDeviceSynchronize();

  hipblasDestroy(handle);
}

/**
 * @brief Make the principal vectors of the data whose covariance matrix is
 * given as a component of args.
 *
 * Principal vectors, i.e., eigenvectors of the covariance matrix, are stored
 * in place of the covariance matrix in column major order.
 *
 * @param args Covariance matrix, number of features.
 */
void make_principal_vectors(MakePVArgs &args) {
  auto m = args.n_feats; // number of rows/columns
  auto lda = args.n_feats; // leading dimension of covariance matrix
  auto n_pcs = args.n_pcs == 0 ? m : std::min(args.n_pcs, m);

  float *eigvecs = thrust::raw_pointer_cast(args.cov_matrix.data());
  float *eigvals = nullptr;
  float *workspace = nullptr;
  auto lwork = 0;
  int *devInfo = nullptr;

  hipMallocManaged(&eigvals, m * sizeof(double));
  hipMallocManaged(&devInfo, sizeof(int));
  hipsolverHandle_t handle = nullptr;

  // create a handle to cuSolverDN library context
  hipsolverStatus_t cusolver_status = hipsolverDnCreate(&handle);
  assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

  // don't throw out the eigenvectors
  hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR;
  hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
  cusolver_status = hipsolverDnSsyevd_bufferSize(handle, jobz, uplo, m, eigvecs,
                                                lda, eigvals, &lwork);
  assert(cusolver_status == HIPSOLVER_STATUS_SUCCESS);

  // compute the eigenvalues and eigenvectors
  hipMallocManaged(&workspace, lwork * sizeof(double));
  cusolver_status = hipsolverDnSsyevd(handle, jobz, uplo, m, eigvecs, lda, eigvals,
                                     workspace, lwork, devInfo);

  hipError_t cuda_status = hipDeviceSynchronize();
  assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
  assert(hipSuccess == cuda_status);

  // truncate the eigenvector matrix to just the number of desired principal
  // vectors
  args.cov_matrix.resize(m * n_pcs);

  hipFree(devInfo);
  hipFree(eigvals);
  hipFree(workspace);
  if (handle) {
    hipsolverDnDestroy(handle);
  }
}

/**
 * @brief Project features onto principal vectors.
 * @param args
 */
void project_onto_pvs(ProjectOntoPVsArgs &args) {
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  auto n_pcs = args.n_pcs;
  auto n_obs = args.n_obs;
  auto n_feats = args.n_feats;

  float *pvs = thrust::raw_pointer_cast(args.pvs.data());
  float *observations = thrust::raw_pointer_cast(args.observations.data());
  float *projections = thrust::raw_pointer_cast(args.projections.data());

  auto alpha = 1.0f;
  auto beta = 0.f;

  hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
              n_pcs, n_obs, n_feats,
              &alpha,
              pvs, n_feats, observations, n_feats,
              &beta,
              projections, n_pcs);

  hipDeviceSynchronize();

  hipblasDestroy(handle);
}
